#include <stdio.h>

#include <hip/hip_runtime.h>

#include "./nj_read/nj_read.cuh"

#include "./nj/nj_flex.cuh"
#include "./nj/nj_normal.cuh"

#include "./time_analisys.cuh"

int main(int argc, char const *argv[])
{

    if (argc != 6)
    {
        printf("Arguments in the form: [file] [type] [p] [k] [TPB]\n");
        return 1;
    }

    const char *file = argv[1];
    int type = atoi(argv[2]);
    float p_value = atof(argv[3]);
    int k_number = atoi(argv[4]);
    int TPB = atoi(argv[5]);

    printf("%s, %d, %.3f, %d\n", file, type, p_value, k_number);

    nj_read_t read;
    nj_data_t data;
    nj_read_init(&read);

    nj_read_file(&read, file);

    data = nj_data_to_device(read, p_value, k_number);

    time_start();

    if (type == 0) // NJ
    {
        nj_normal(data, TPB);
    }
    else if (type == 1) // FNJ - Reduce
    {
        // TODO: Function call
        nj_flex(data, TPB);
    }
    else if (type == 2) // FNJ - kHeap
    {
        // TODO: Function call
    }

    time_end();

    printf("%d; %.4f;\n", read.N, elapsed_time);

    free_nj_data_device(data);
    free_nj_read(read);

    return 0;
}
